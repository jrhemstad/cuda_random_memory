
#include <benchmark/benchmark.h>

#include <cuda/atomic>
#include <synchronization.hpp>
#include <sys/types.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/random.h>
#include <thrust/transform.h>


template <typename T> void BM_weak_sequential_load(benchmark::State &state) {
  thrust::device_vector<T> input(state.range(0));
  thrust::device_vector<T> output(input.size());
  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(thrust::device, input.begin(), input.end(),
                      output.begin(), [] __device__(auto v) { return v; });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

/*
BENCHMARK_TEMPLATE(BM_weak_sequential_load, int32_t)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename T> void BM_weak_random_load(benchmark::State &state) {
  thrust::device_vector<T> input(state.range(0));
  thrust::device_vector<T> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<std::size_t>(0), l);

  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};

    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get()] __device__(auto random_index) {
          return input_data[random_index];
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

/*
BENCHMARK_TEMPLATE(BM_weak_random_load, int32_t)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename U> using Atomic = cuda::atomic<U, cuda::thread_scope_device>;

template <typename T, cuda::std::memory_order mem_order>
void BM_atomic_sequential_load(benchmark::State &state) {

  thrust::device_vector<Atomic<T>> input(state.range(0));
  thrust::device_vector<T> output(input.size());

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(thrust::device, input.cbegin(), input.cend(),
                      output.begin(), [] __device__(auto const &atomic_value) {
                        return atomic_value.load(mem_order);
                      });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

/*
BENCHMARK_TEMPLATE(BM_atomic_sequential_load, int32_t,
                   cuda::std::memory_order_relaxed)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(BM_atomic_sequential_load, int32_t,
                   cuda::std::memory_order_seq_cst)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename T, cuda::std::memory_order mem_order>
void BM_weak_probing_load(benchmark::State &state) {
  thrust::device_vector<T> input(state.range(0));
  thrust::device_vector<T> output(input.size());
  
  
  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<std::size_t>(0), l);

  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get(), size = input.size()] __device__(auto random_index) {
          for(auto i = 0; i < 5; ++i) {
            volatile auto k = input_data[random_index];
            random_index = (random_index + 1) % size;
          }
          return input_data[random_index];
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

BENCHMARK_TEMPLATE(BM_weak_probing_load, int32_t,
                   cuda::std::memory_order_relaxed)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(BM_weak_probing_load, int32_t,
                   cuda::std::memory_order_seq_cst)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


template <typename T, cuda::std::memory_order mem_order>
void BM_atomic_random_load(benchmark::State &state) {
  thrust::device_vector<Atomic<T>> input(state.range(0));
  thrust::device_vector<T> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);

  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get()] __device__(auto random_index) {
          return input_data[random_index].load(mem_order);
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

/*
BENCHMARK_TEMPLATE(BM_atomic_random_load, int32_t,
                   cuda::std::memory_order_relaxed)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(BM_atomic_random_load, int32_t,
                   cuda::std::memory_order_seq_cst)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename T, cuda::std::memory_order mem_order>
void BM_atomic_probing_load(benchmark::State &state) {
  thrust::device_vector<Atomic<T>> input(state.range(0));
  thrust::device_vector<T> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);

  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get(), size = input.size()] __device__(auto random_index) {
          for(auto i = 0; i < 5; ++i) {
            volatile auto k = input_data[random_index].load(mem_order);
            random_index = (random_index + 1) % size;
          }
          return input_data[random_index].load(mem_order);
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 * sizeof(T));
}

BENCHMARK_TEMPLATE(BM_atomic_probing_load, int32_t,
                   cuda::std::memory_order_relaxed)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(BM_atomic_probing_load, int32_t,
                   cuda::std::memory_order_seq_cst)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);