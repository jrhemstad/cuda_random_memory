
#include <atomic>
#include <benchmark/benchmark.h>

#include <cuda/atomic>
#include <synchronization.hpp>
#include <sys/types.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/pair.h>
#include <thrust/random.h>

template <typename First, typename Second>
void BM_weak_sequential(benchmark::State &state) {

  thrust::device_vector<thrust::pair<First, Second>> input(state.range(0));
  thrust::device_vector<thrust::pair<First, Second>> output(input.size());

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(thrust::device, input.cbegin(), input.cend(),
                      output.begin(),
                      [] __device__(auto const &p) { return p; });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}

/*
BENCHMARK_TEMPLATE(BM_weak_sequential, int32_t, int32_t)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename First, typename Second>
void BM_weak_random(benchmark::State &state) {

  thrust::device_vector<thrust::pair<First, Second>> input(state.range(0));
  thrust::device_vector<thrust::pair<First, Second>> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);
  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get()] __device__(auto random_index) {
          return input_data[random_index];
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}

/*
BENCHMARK_TEMPLATE(BM_weak_random, int32_t, int32_t)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename First, typename Second>
void BM_weak_random_probing(benchmark::State &state) {

  thrust::device_vector<thrust::pair<First, Second>> input(state.range(0));
  thrust::device_vector<First> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);
  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get(), size = input.size()] __device__(auto random_index) {
          uint32_t count = 0;
          for(auto i = 0; i < 4; ++i) {
            count += input_data[random_index].first;
            random_index = (random_index + 1) % size;
          }
          count += input_data[random_index].first;
          count += input_data[random_index].second;
          return count;
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}

BENCHMARK_TEMPLATE(BM_weak_random_probing, int32_t, int32_t)
    ->RangeMultiplier(10)
    ->Range(100'000'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);


template <typename T> using Atomic = cuda::atomic<T, cuda::thread_scope_device>;

static constexpr auto RELAXED{cuda::std::memory_order_relaxed};
static constexpr auto SEQCST{cuda::std::memory_order_seq_cst};

template <typename First, typename Second, cuda::std::memory_order F_mem_order,
          cuda::std::memory_order S_mem_order>
void BM_atomic_sequential(benchmark::State &state) {

  thrust::device_vector<thrust::pair<Atomic<First>, Atomic<Second>>> input(
      state.range(0));
  thrust::device_vector<thrust::pair<First, Second>> output(input.size());

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(thrust::device, input.cbegin(), input.cend(),
                      output.begin(), [] __device__(auto const &p) {
                        auto const f = p.first.load(F_mem_order);
                        auto const s = p.second.load(S_mem_order);
                        return thrust::make_pair(f, s);
                      });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}
/*
BENCHMARK_TEMPLATE(BM_atomic_sequential, int32_t, int32_t, RELAXED, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_atomic_sequential, int32_t, int32_t, SEQCST, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_atomic_sequential, int32_t, int32_t, RELAXED, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_atomic_sequential, int32_t, int32_t, SEQCST, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000, 1'000'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename First, typename Second, cuda::std::memory_order F_mem_order,
          cuda::std::memory_order S_mem_order>
void BM_atomic_random(benchmark::State &state) {

  thrust::device_vector<thrust::pair<Atomic<First>, Atomic<Second>>> input(
      state.range(0));
  thrust::device_vector<thrust::pair<First, Second>> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);
  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get()] __device__(auto random_index) {
          auto const &p = input_data[random_index];
          auto const f = p.first.load(F_mem_order);
          auto const s = p.second.load(S_mem_order);
          return thrust::make_pair(f, s);
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}
/*
BENCHMARK_TEMPLATE(BM_atomic_random, int32_t, int32_t, RELAXED, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random, int32_t, int32_t, SEQCST, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random, int32_t, int32_t, RELAXED, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random, int32_t, int32_t, SEQCST, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
*/

template <typename First, typename Second, cuda::std::memory_order F_mem_order,
          cuda::std::memory_order S_mem_order>
void BM_atomic_random_probing(benchmark::State &state) {

  thrust::device_vector<thrust::pair<Atomic<First>, Atomic<Second>>> input(
      state.range(0));
  thrust::device_vector<First> output(input.size());

  auto l = [input_size = input.size()] __device__(auto i) {
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int32_t> dist(0, input_size);
    rng.discard(i);
    return dist(rng);
  };

  auto const random_begin = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), l);
  auto const random_end = random_begin + input.size();

  for (auto _ : state) {
    cuda_event_timer raii{state};
    thrust::transform(
        thrust::device, random_begin, random_end, output.begin(),
        [input_data = input.data().get(), size = input.size()] __device__(auto random_index) {
          
          uint32_t count = 0;

          for(auto i = 0; i < 4; ++i) {
            count += input_data[random_index].first.load(F_mem_order);
            random_index = (random_index + 1) % size;
          }
          count += input_data[random_index].first.load(F_mem_order);
          count += input_data[random_index].second.load(S_mem_order);
          return count;
        });
  }
  state.SetBytesProcessed(int64_t(state.iterations()) *
                          int64_t(state.range(0)) * 2 *
                          sizeof(thrust::pair<First, Second>));
}

BENCHMARK_TEMPLATE(BM_atomic_random_probing, int32_t, int32_t, RELAXED, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random_probing, int32_t, int32_t, SEQCST, RELAXED)
    ->RangeMultiplier(10)
    ->Range(100'000'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random_probing, int32_t, int32_t, RELAXED, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
BENCHMARK_TEMPLATE(BM_atomic_random_probing, int32_t, int32_t, SEQCST, SEQCST)
    ->RangeMultiplier(10)
    ->Range(100'000'000, 100'000'000)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

/*

template <typename T> void BM_weak_random_load(benchmark::State &state) {
thrust::device_vector<T> v(state.range(0));

auto l = [input_size = v.size()] __device__(auto i) {
thrust::default_random_engine rng;
thrust::uniform_int_distribution<int32_t> dist(0, input_size);
rng.discard(i);
return dist(rng);
};

auto const begin = thrust::make_transform_iterator(
  thrust::make_counting_iterator<int32_t>(0), l);

auto const end = thrust::make_transform_iterator(
  thrust::make_counting_iterator<int32_t>(v.size()), l);

for (auto _ : state) {
cuda_event_timer raii{state};
thrust::for_each(thrust::device, begin, end,
                 [input_data = v.data().get()] __device__(auto index) {
                   volatile auto l = input_data[index];
                 });
}
state.SetBytesProcessed(int64_t(state.iterations()) *
                      int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_weak_random_load, int32_t)
->RangeMultiplier(10)
->Range(100'000, 1'000'000'000)
->UseManualTime()
->Unit(benchmark::kMillisecond);

template <typename T, cuda::std::memory_order mem_order>
void BM_atomic_sequential_load(benchmark::State &state) {
thrust::device_vector<cuda::atomic<T, cuda::thread_scope_device>> v(
  state.range(0));
for (auto _ : state) {
cuda_event_timer raii{state};
auto const begin = thrust::make_counting_iterator(0);
auto const end = begin + state.range(0);
thrust::for_each(thrust::device, begin, end,
                 [input_data = v.data().get(),
                  input_size = v.size()] __device__(auto index) {
                   volatile auto l = input_data[index].load(mem_order);
                 });
}
state.SetBytesProcessed(int64_t(state.iterations()) *
                      int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_atomic_sequential_load, int32_t,
               cuda::std::memory_order_relaxed)
->RangeMultiplier(10)
->Range(100'000, 1'000'000'000)
->UseManualTime()
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_atomic_sequential_load, int32_t,
               cuda::std::memory_order_seq_cst)
->RangeMultiplier(10)
->Range(100'000, 1'000'000'000)
->UseManualTime()
->Unit(benchmark::kMillisecond);

template <typename T, cuda::std::memory_order mem_order>
void BM_atomic_random_load(benchmark::State &state) {
thrust::device_vector<cuda::atomic<T, cuda::thread_scope_device>> v(
  state.range(0));

auto l = [input_size = v.size()] __device__(auto i) {
thrust::default_random_engine rng;
thrust::uniform_int_distribution<int32_t> dist(0, input_size);
rng.discard(i);
return dist(rng);
};

auto const begin = thrust::make_transform_iterator(
  thrust::make_counting_iterator<int32_t>(0), l);

auto const end = thrust::make_transform_iterator(
  thrust::make_counting_iterator<int32_t>(v.size()), l);

for (auto _ : state) {
cuda_event_timer raii{state};
thrust::for_each(thrust::device, begin, end,
                 [input_data = v.data().get()] __device__(auto index) {
                   volatile auto l = input_data[index].load(mem_order);
                 });
}
state.SetBytesProcessed(int64_t(state.iterations()) *
                      int64_t(state.range(0)) * sizeof(T));
}
BENCHMARK_TEMPLATE(BM_atomic_random_load, int32_t,
               cuda::std::memory_order_relaxed)
->RangeMultiplier(10)
->Range(100'000, 1'000'000'000)
->UseManualTime()
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_atomic_random_load, int32_t,
               cuda::std::memory_order_seq_cst)
->RangeMultiplier(10)
->Range(100'000, 1'000'000'000)
->UseManualTime()
->Unit(benchmark::kMillisecond);
*/
