#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>
#include <cstdint>
#include <cuda/atomic>
#include <synchronization.hpp>
#include <sys/types.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include "hash_functions.cuh"

auto constexpr RELAXED = cuda::std::memory_order_relaxed;
auto constexpr SEQCST = cuda::std::memory_order_seq_cst;

template <typename T> using Atomic = cuda::atomic<T, cuda::thread_scope_device>;

template <typename K, typename V>
using Slot = thrust::pair<Atomic<K>, Atomic<V>>;

template <cuda::std::memory_order K_mem_order,
          cuda::std::memory_order V_mem_order, std::size_t block_size,
          typename K, typename V>
__global__ void find(Slot<K, V> const *slots, std::size_t num_slots, K const *k,
                     V *output, std::size_t num_keys, K empty_key,
                     V empty_value) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ V buffer[block_size];
  if (tid < num_keys) {
    auto const my_key = k[tid];
    auto const key_hash = MurmurHash3_32<K>{}(my_key);
    auto slot_index = key_hash % num_slots;
    while (true) {
      auto const current_slot = &slots[slot_index];
      auto const existing_key = current_slot->first.load(K_mem_order);

      // Matching key
      if (existing_key == my_key) {
        // output[tid] = current_slot->second.load(V_mem_order);
        buffer[threadIdx.x] = current_slot->second.load(V_mem_order);
        break;
      }
      // Empty slot. Key doesn't exist
      if (existing_key == empty_key) {
        // output[tid] = empty_value;
        buffer[threadIdx.x] = empty_value;
        break;
      }
      slot_index = (slot_index + 1) % num_slots;
    }
  }
  __syncthreads();

  if (tid < num_keys) {
    output[tid] = buffer[threadIdx.x];
  }
}

/**
 * @brief Generates input sizes and hash table occupancies
 *
 */
static void generate_size_and_occupancy(benchmark::internal::Benchmark *b) {
  for (auto occupancy = 40; occupancy <= 90; occupancy += 10) {
    for (auto size = 100'000'000; size <= 100'000'000; size *= 10) {
      b->Args({size, occupancy});
    }
  }
}

template <typename K, typename V, cuda::std::memory_order K_mem_order,
          cuda::std::memory_order V_mem_order>
void BM_map_proxy(benchmark::State &state) {
  auto const num_keys = state.range(0);
  auto const occupancy = (state.range(1) / double{100});
  auto const num_slots = static_cast<std::size_t>(num_keys / occupancy);
  thrust::device_vector<Slot<K, V>> slots(num_slots);

  constexpr K empty_key{-1};
  constexpr V empty_value{-1};

  // Initialize slots to empty
  thrust::for_each(thrust::device, slots.begin(), slots.end(),
                   [] __device__(auto &slot) {
                     new (&slot.first) Atomic<K>{empty_key};
                     new (&slot.second) Atomic<V>{empty_value};
                   });

  thrust::device_vector<K> keys(num_keys);
  thrust::sequence(keys.begin(), keys.end(), 0);
  auto values = thrust::make_counting_iterator(0);
  auto kvs =
      thrust::make_zip_iterator(thrust::make_tuple(keys.begin(), values));

  // Insert the key/value pairs into the map
  thrust::for_each(
      thrust::device, kvs, kvs + num_keys,
      [s = slots.data().get(),
       num_slots = slots.size()] __device__(auto const &kv) {
        auto const k = thrust::get<0>(kv);
        auto const v = thrust::get<1>(kv);
        auto const key_hash = MurmurHash3_32<K>{}(k);
        auto slot_index = key_hash % num_slots;

        while (true) {
          auto const current_slot = &s[slot_index];

          auto &slot_key = current_slot->first;
          auto &slot_value = current_slot->second;

          auto expected_key = empty_key;
          auto expected_value = empty_value;

          bool const key_success = slot_key.compare_exchange_strong(
              expected_key, k, cuda::std::memory_order_relaxed);

          bool value_success = slot_value.compare_exchange_strong(
              expected_value, v, cuda::std::memory_order_relaxed);

          // Usually, both will succeed. Otherwise, whoever won the key CAS is
          // guaranteed to eventually update the value
          if (key_success) {
            // If key succeeds and value doesn't, someone else won the value CAS
            // Spin trying to update the value
            while (not value_success) {
              value_success = slot_value.compare_exchange_strong(
                  expected_value = empty_value, v,
                  cuda::std::memory_order_relaxed);
            }
            return;
          } else if (value_success) {
            // Key CAS failed, but value succeeded. Restore the value to it's
            // initial value
            slot_value.store(empty_value, cuda::std::memory_order_relaxed);
          }

          slot_index = (slot_index + 1) % num_slots;
        }
      });

  thrust::device_vector<V> output_values(num_keys);

  for (auto _ : state) {
    cuda_event_timer raii{state};
    constexpr auto block_size{128};
    auto grid_size = (num_keys + block_size - 1) / block_size;
    find<K_mem_order, V_mem_order, block_size><<<grid_size, block_size>>>(
        slots.data().get(), num_slots, keys.data().get(),
        output_values.data().get(), num_keys, empty_key, empty_value);
  }

  state.SetBytesProcessed((sizeof(K) + sizeof(V)) *
                          int64_t(state.iterations()) *
                          int64_t(state.range(0)));
}

BENCHMARK_TEMPLATE(BM_map_proxy, int32_t, int32_t, RELAXED, RELAXED)
    ->UseManualTime()
    ->Apply(generate_size_and_occupancy)
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_map_proxy, int32_t, int32_t, SEQCST, RELAXED)
    ->UseManualTime()
    ->Apply(generate_size_and_occupancy)
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_map_proxy, int32_t, int32_t, RELAXED, SEQCST)
    ->UseManualTime()
    ->Apply(generate_size_and_occupancy)
    ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(BM_map_proxy, int32_t, int32_t, SEQCST, SEQCST)
    ->UseManualTime()
    ->Apply(generate_size_and_occupancy)
    ->Unit(benchmark::kMillisecond);
